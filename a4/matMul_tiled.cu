#include "hip/hip_runtime.h"
#include "matMul.h"

__global__ static void matrixMulCuda(const uint32_t n, const float *dev_A, const float *dev_B,
            float *dev_C)
{
    extern __shared__ int s[]; //shared memory allocated during kernel launch

    // create A_tile and B_tile from shared memory s
    float *A_tile = (float *)s;
    float *B_tile = A_tile + blockDim.x*blockDim.y;

    float partial = 0.0;

    // blcok indxes
    uint16_t bx = blockIdx.x; uint16_t by = blockIdx.y;
    uint16_t bdx = blockDim.x; uint16_t bdy = blockDim.y;
    uint16_t tx = threadIdx.x; uint16_t ty = threadIdx.y;

    // tile
    uint16_t numTiles = n/bdx; // should also equal n/blockDim.y

    // which col and row each thread has of C
    uint16_t row = by*bdy + ty;
    uint16_t col = bx*bdx + tx;


    for (uint16_t m = 0; m < numTiles; m++) {
        // Loads are coallesced
        //A_tile[ty][tx] = dev_A[row][m*bdx + tx];
        A_tile[bdx*ty + tx] = dev_A[n*row + m*bdx + tx];


        //B_tile[ty][tx] = dev_B[m*bdy + ty][col];
        B_tile[bdy*ty + tx] = dev_B[n*(m*bdy + ty) + col];

        // wait for all threads to finish
        __syncthreads();

        // compute partial dot product
        for (uint16_t x = 0; x < bdx; x++)
            partial += A_tile[bdx*ty + x] * B_tile[bdy*x + tx];

        __syncthreads();
    }

    // update global memory
    dev_C[n*row + col] = partial;
}


void matMul_tiled(const uint32_t n, const float *A, const float *B, float *C, const uint32_t block_dim)
{
    float *dev_A, *dev_B, *dev_C;
    // copy A
    hipMalloc(&dev_A, sizeof(float[n][n]));
    hipMemcpy(dev_A, A, sizeof(float[n][n]), hipMemcpyHostToDevice);

    // Copy B
    hipMalloc(&dev_B, sizeof(float[n][n]));
    hipMemcpy(dev_B, (float*)B, sizeof(float[n][n]), hipMemcpyHostToDevice);

    // Allocate space for C
    hipMalloc(&dev_C, sizeof(float[n][n]));

    dim3 Block(block_dim, block_dim);
    dim3 Grid(n/Block.x, n/Block.y);

    matrixMulCuda<<< Grid, Block, 2*sizeof(float[Block.y][Block.x])>>>(n, dev_A, dev_B, dev_C);

    hipMemcpy(C, dev_C, sizeof(float[n][n]), hipMemcpyDeviceToHost);

    // Clean up memory
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

}
